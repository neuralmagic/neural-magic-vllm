#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <assert.h>

#include "../../dispatch_utils.h"

static inline __device__ int8_t float_to_int8_rn(float x)
{
#ifdef USE_ROCM
    float dst;
    // Round to nearest even
    asm volatile("v_rndne_f32 %0, %1;" : "=v"(dst) : "v"(x));
    // Saturate
    dst = dst < -128.0f ? -128.0f : dst;
    dst = dst > 127.0f ? 127.0f : dst;
    return static_cast<int8_t>(dst);
#else
    uint32_t dst;
    asm volatile("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(dst) : "f"(x));
    return reinterpret_cast<const int8_t&>(dst);
#endif
}

namespace vllm {

// TODO (varun) : Merge this into reduction utils and use the existing interface
// TODO (varun) : Add unit tests for this
template<typename T>
__inline__ __device__ T warpReduceMax(T val)
{
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1)
        val = max(val, __shfl_xor_sync(0xffffffff, val, mask, 32));
    return val;
}

/* Calculate the maximum of all elements in a block */
template<typename T>
__inline__ __device__ T blockReduceMax(T val)
{
    static __shared__ T shared[32];
    int                 lane = threadIdx.x & 0x1f;  // in-warp idx
    int                 wid  = threadIdx.x >> 5;    // warp idx
    val = warpReduceMax(val);  // get maxx in each warp
    if (lane == 0)  // record in-warp maxx by warp Idx
        shared[wid] = val;
    __syncthreads();
    val = (threadIdx.x < (blockDim.x / 32.f)) ? shared[lane] : -1e20f;
    val = warpReduceMax(val);
    return val;
}

template <typename scalar_t, typename scale_type>
__global__ void static_scaled_int8_quant_kernel(
  const scalar_t* __restrict__ input,
  int8_t* __restrict__ out,
  scale_type scale,
  const int hidden_size) {
  const int tid = threadIdx.x;
  const int token_idx = blockIdx.x;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] =
        float_to_int8_rn(((float)input[token_idx * hidden_size + i]) / scale);
  }
}

template <typename scalar_t, typename scale_type>
__global__ void quant_per_token_kernel(
  const scalar_t* __restrict__ input,
  int8_t* __restrict__ out,
  scale_type scale,
  const int hidden_size) {

  const int tid = threadIdx.x;
  const int token_idx = blockIdx.x;

  float amax_val = 0.0f;
  const float zero = 0.0f;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    float val = (float)input[token_idx * hidden_size + i];
    val = val > zero ? val : -val;
    if (val > amax_val)
      amax_val = val;
  }

  __shared__ float s_amax;
  const float block_amax_val = blockReduceMax(amax_val);
  if (tid == 0) {
    s_amax = block_amax_val;
    scale[token_idx] = block_amax_val / 127.0f;
  }
  __syncthreads();

  float tmp_scale = 127.0f / s_amax;
  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] =
        float_to_int8_rn(((float)input[token_idx * hidden_size + i]) * tmp_scale);
  }
}

} // namespace vllm

void static_scaled_int8_quant(
  torch::Tensor& out,   // [..., hidden_size]
  torch::Tensor& input, // [..., hidden_size]
  float scale) {
  assert(input.is_contiguous());
  assert(out.is_contiguous());
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "static_scaled_int8_quant_kernel", [&] {
    vllm::static_scaled_int8_quant_kernel<scalar_t, float><<<grid, block, 0, stream>>>(
      input.data_ptr<scalar_t>(),
      out.data_ptr<int8_t>(),
      scale,
      hidden_size);
  });
}

void quant_per_token(
  torch::Tensor& out,   // [..., hidden_size]
  torch::Tensor& input, // [..., hidden_size]
  torch::Tensor& scales) {
  assert(input.is_contiguous());
  assert(out.is_contiguous());
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "quant_per_token_kernel", [&] {
    vllm::quant_per_token_kernel<scalar_t, float*><<<grid, block, 0, stream>>>(
      input.data_ptr<scalar_t>(),
      out.data_ptr<int8_t>(),
      scales.data_ptr<float>(),
      hidden_size);
  });
}

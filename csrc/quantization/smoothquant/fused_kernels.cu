#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <assert.h>

#include "../../dispatch_utils.h"
#include "../../reduction_utils.cuh"
#include "quant_utils.cuh"

namespace vllm {

template <typename scalar_t, typename scale_type, bool use_per_token_quant>
__global__ void quant_kernel(
  const scalar_t* __restrict__ input,
  int8_t* __restrict__ out,
  scale_type scale,
  const int hidden_size) {
  const int tid = threadIdx.x;
  const int token_idx = blockIdx.x;

  if constexpr (use_per_token_quant) {
    float amax_val = 0.0f;
    const float zero = 0.0f;

    for (int i = tid; i < hidden_size; i += blockDim.x) {
      float val = (float)input[token_idx * hidden_size + i];
      val = val > zero ? val : -val;
      if (val > amax_val)
        amax_val = val;
    }

    __shared__ float s_amax;
    const float block_amax_val = blockReduceMax(amax_val);
    if (tid == 0) {
      s_amax = block_amax_val;
      scale[token_idx] = block_amax_val / 127.0f;
    }
    __syncthreads();

    float tmp_scale = 127.0f / s_amax;
    for (int i = tid; i < hidden_size; i += blockDim.x) {
      out[token_idx * hidden_size + i] =
          float_to_int8_rn(((float)input[token_idx * hidden_size + i]) * tmp_scale);
    }
  } else {
    for (int i = tid; i < hidden_size; i += blockDim.x) {
      out[token_idx * hidden_size + i] =
          float_to_int8_rn(((float)input[token_idx * hidden_size + i]) / scale);
    }
  }
}
} // namespace vllm

void quant(
  torch::Tensor& out,   // [..., hidden_size]
  torch::Tensor& input, // [..., hidden_size]
  float scale) {
  assert(input.is_contiguous());
  assert(out.is_contiguous());
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "quant_kernel", [&] {
    vllm::quant_kernel<scalar_t, float, false><<<grid, block, 0, stream>>>(
      input.data_ptr<scalar_t>(),
      out.data_ptr<int8_t>(),
      scale,
      hidden_size);
  });
}

void quant(
  torch::Tensor& out,   // [..., hidden_size]
  torch::Tensor& input, // [..., hidden_size]
  torch::Tensor& scale) { // [num_tokens]
  assert(input.is_contiguous());
  assert(out.is_contiguous());
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "quant_kernel", [&] {
    vllm::quant_kernel<scalar_t, float*, true><<<grid, block, 0, stream>>>(
      input.data_ptr<scalar_t>(),
      out.data_ptr<int8_t>(),
      scale.data_ptr<float>(),
      hidden_size);
  });
}
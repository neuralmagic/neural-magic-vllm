#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include "../../dispatch_utils.h"
#include "../../reduction_utils.cuh"
// #include "quant_utils.cuh"

namespace vllm {

static inline __device__ int8_t float_to_int8_rn(float x) {
  uint32_t dst;
  asm volatile("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int8_t &>(dst);
}

template <typename scalar_t>
__global__ void rms_norm_quant_kernel(
  int8_t* __restrict__ out,         // [..., hidden_size]
  const scalar_t* __restrict__ input, // [..., hidden_size]
  float* __restrict__ tmp, // [..., hidden_size]
  const scalar_t* __restrict__ weight, // [hidden_size]
  float* __restrict__ scale, // [num_tokens]
  const double epsilon,
  const int num_tokens,
  const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    const float x = (float) input[blockIdx.x * hidden_size + idx];
    variance += x * x;
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  __shared__ float s_amax;
  float amax_val = 0.0f;
  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float) input[blockIdx.x * hidden_size + idx];
    x = x * s_variance * (float) (weight[idx]);
    // input[blockIdx.x * hidden_size + idx] = (scalar_t) x;
    tmp[blockIdx.x * hidden_size + idx] = x;
    amax_val = fmaxf(amax_val, fabsf(x));
  }
  amax_val = blockReduceMax(amax_val);
  if (threadIdx.x == 0) {
    s_amax = amax_val;
    scale[blockIdx.x] = amax_val / 127.0f;
  }
  __syncthreads();

  float tmp_scale = 127.0f / s_amax;
  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    // out[blockIdx.x * hidden_size + idx] = 
    //     float_to_int8_rn(((float) input[blockIdx.x * hidden_size + idx]) * tmp_scale);
    out[blockIdx.x * hidden_size + idx] = 
        float_to_int8_rn((tmp[blockIdx.x * hidden_size + idx]) * tmp_scale);
  }
}


template<typename scalar_t>
__global__ void add_residual_rms_norm_quant_kernel(
  int8_t* __restrict__ out,             // [..., hidden_size]
  const scalar_t* __restrict__ input,           // [..., hidden_size]
  scalar_t* __restrict__ residual,        // [..., hidden_size]
  float* __restrict__ tmp,                // [..., hidden_size]
  const scalar_t* __restrict__ weight,    // [hidden_size]
  float* __restrict__ scale,             // [num_tokens]
  const float epsilon,
  const int num_tokens,
  const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    scalar_t z = input[blockIdx.x * hidden_size + idx];
    z += residual[blockIdx.x * hidden_size + idx];
    float x = (float) z;
    variance += x * x;
    residual[blockIdx.x * hidden_size + idx] = z;
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  __shared__ float s_amax;
  float amax_val = 0.0f;
  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float) residual[blockIdx.x * hidden_size + idx];
    x = x * s_variance * (float) (weight[idx]);
    // [blockIdx.x * hidden_size + idx] = (scalar_t) x;
    tmp[blockIdx.x * hidden_size + idx] = x;
    amax_val = fmaxf(amax_val, fabsf(x));
  }
  amax_val = blockReduceMax(amax_val);
  if (threadIdx.x == 0) {
    s_amax = amax_val;
    scale[blockIdx.x] = amax_val / 127.0f;
  }
  __syncthreads();

  float tmp_scale = 127.0f / s_amax;
  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    // out[blockIdx.x * hidden_size + idx] = 
    //     float_to_int8_rn(((float) input[blockIdx.x * hidden_size + idx]) * tmp_scale);
    out[blockIdx.x * hidden_size + idx] = 
        float_to_int8_rn((tmp[blockIdx.x * hidden_size + idx]) * tmp_scale);
  }
}

// template <typename scalar_t>
// __global__ void quant_kernel(
//   const scalar_t* __restrict__ input,
//   int8_t* __restrict__ out,
//   float* __restrict__ scale,
//   const int num_tokens,
//   const int hidden_size) {
//   __shared__ float s_amax;
//   float amax_val = 0.0f;

//   for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
//     float x = (float) input[blockIdx.x * hidden_size + idx];
//     amax_val = fmaxf(amax_val, fabsf(x));
//   }
//   amax_val = blockReduceMax(amax_val);
//   if (threadIdx.x == 0) {
//     s_amax = amax_val;
//     scale[blockIdx.x] = amax_val / 127.0f;
//   }
//   __syncthreads();

//   float tmp_scale = 127.0f / s_amax;
//   for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
//     out[blockIdx.x * hidden_size + idx] = 
//         float_to_int8_rn(((float) input[blockIdx.x * hidden_size + idx]) * tmp_scale);
//   }
// }

} // namespace vllm

void rms_norm_quant(
  torch::Tensor& out,    // [..., hidden_size]
  torch::Tensor const& input,  // [..., hidden_size]
  torch::Tensor& tmp,    // [..., hidden_size]
  torch::Tensor const& weight, // [hidden_size]
  torch::Tensor& scale, // [num_tokens]
  double const epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_quant_kernel", [&] {
    vllm::rms_norm_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
      out.data_ptr<int8_t>(),
      input.data_ptr<scalar_t>(),
      tmp.data_ptr<float>(),
      weight.data_ptr<scalar_t>(),
      scale.data_ptr<float>(),
      epsilon,
      num_tokens,
      hidden_size);
  });
}

void add_residual_rms_norm_quant(
  torch::Tensor& out,      // [..., hidden_size]
  torch::Tensor const& input,    // [..., hidden_size]
  torch::Tensor& residual, // [..., hidden_size]
  torch::Tensor& tmp,      // [..., hidden_size]
  torch::Tensor const& weight,   // [hidden_size]
  torch::Tensor& scale,    // [num_tokens]
  double const epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
    input.scalar_type(),
    "add_residual_rms_norm_quant_kernel", [&] {
      vllm::add_residual_rms_norm_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<int8_t>(),
        input.data_ptr<scalar_t>(),
        residual.data_ptr<scalar_t>(),
        tmp.data_ptr<float>(),
        weight.data_ptr<scalar_t>(),
        scale.data_ptr<float>(),
        epsilon,
        num_tokens,
        hidden_size);
    });
}

// void quant(
//   torch::Tensor& out,   // [..., hidden_size]
//   torch::Tensor& input, // [..., hidden_size]
//   torch::Tensor& scale) { // [num_tokens]
//   assert(input.is_contiguous());
//   assert(out.is_contiguous());
//   int hidden_size = input.size(-1);
//   int num_tokens = input.numel() / hidden_size;

//   dim3 grid(num_tokens);
//   dim3 block(std::min(hidden_size, 1024));
//   const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
//   const hipStream_t stream = at::cuda::getCurrentCUDAStream();
//   VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "quant_kernel", [&] {
//     vllm::quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
//       input.data_ptr<scalar_t>(),
//       out.data_ptr<int8_t>(),
//       scale.data_ptr<float>(),
//       num_tokens,
//       hidden_size);
//   });
// }